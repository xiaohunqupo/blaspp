#include "hip/hip_runtime.h"
#include "blas/device.hh"
#include "operators.cuh"

#if defined(BLAS_HAVE_CUBLAS)

namespace blas {

// Each thread conjugates 1 item
template <typename scalar_t>
__global__ void conj_kernel(
    int64_t n,
    scalar_t const* x, int64_t incx, int64_t ix,
    scalar_t*       y, int64_t incy, int64_t iy)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[ i*incy + iy ] = conj_device( x[ i*incx + ix ] );
}

//------------------------------------------------------------------------------
/// Conjugates each element of the vector x and stores in y.
///
/// @param[in] n
///     Number of elements in the vector. n >= 0.
///
/// @param[in] x
///     Pointer to the input vector of length n.
///
/// @param[in] incx
///     Stride between elements of x. incx >= 1.
///
/// @param[out] y
///     Pointer to output vector
///     On exit, each element y[i] is updated as y[i] = conj( x[i] ).
///     y may be the same as x.
///
/// @param[in] incy
///     Stride between elements of y. incy >= 1.
///
/// @param[in] queue
///     BLAS++ queue to execute in.
///
template <typename scalar_t>
void conj(
    int64_t n,
    scalar_t const* x, int64_t incx,
    scalar_t*       y, int64_t incy,
    blas::Queue& queue )
{
    blas_error_if( n < 0 );
    blas_error_if( incx == 0 );
    blas_error_if( incy == 0 );

    if (n == 0)
        return;

    const int64_t BlockSize = 1024;

    int64_t n_threads = min( BlockSize, n );
    int64_t n_blocks = ceildiv(n, n_threads);

    int64_t ix = (incx > 0 ? 0 : (1 - n) * incx);
    int64_t iy = (incy > 0 ? 0 : (1 - n) * incy);

    blas_dev_call( hipSetDevice( queue.device() ) );

    if constexpr (std::is_same_v<scalar_t, std::complex<float>>) {
        conj_kernel<hipComplex><<<n_blocks, n_threads, 0, queue.stream()>>>(
            n, (hipComplex*) x, incx, ix, (hipComplex*) y, incy, iy );
    }
    else if constexpr (std::is_same_v<scalar_t, std::complex<double>>) {
        conj_kernel<hipDoubleComplex><<<n_blocks, n_threads, 0, queue.stream()>>>(
            n, (hipDoubleComplex*) x, incx, ix, (hipDoubleComplex*) y, incy, iy );
    }
    else {
    conj_kernel<<<n_blocks, n_threads, 0, queue.stream()>>>(
        n, x, incx, ix, y, incy, iy );
    }

    blas_dev_call( hipGetLastError() );
}

//------------------------------------------------------------------------------
// Explicit instantiations.
template void conj(
    int64_t n,
    float const* x, int64_t incx,
    float* y, int64_t incy,
    blas::Queue& queue);

template void conj(
    int64_t n,
    double const* x, int64_t incx,
    double* y, int64_t incy,
    blas::Queue& queue);

template void conj(
    int64_t n,
    std::complex<float> const* x, int64_t incx,
    std::complex<float>* y, int64_t incy,
    blas::Queue& queue);

template void conj(
    int64_t n,
    std::complex<double> const* x, int64_t incx,
    std::complex<double>* y, int64_t incy,
    blas::Queue& queue);

} // namespace blas

#endif // BLAS_HAVE_CUBLAS
